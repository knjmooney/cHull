#include "hip/hip_runtime.h"
/*
Author: Cao Thanh Tung
Date: 21/01/2010

File Name: pba2DHost.cu

===============================================================================

Copyright (c) 2010, School of Computing, National University of Singapore. 
All rights reserved.

Project homepage: http://www.comp.nus.edu.sg/~tants/pba.html

If you use PBA and you like it or have comments on its usefulness etc., we 
would love to hear from you at <tants@comp.nus.edu.sg>. You may share with us
your experience and any possibilities that we may improve the work/code.

===============================================================================

Redistribution and use in source and binary forms, with or without modification,
are permitted provided that the following conditions are met:

Redistributions of source code must retain the above copyright notice, this list of
conditions and the following disclaimer. Redistributions in binary form must reproduce
the above copyright notice, this list of conditions and the following disclaimer
in the documentation and/or other materials provided with the distribution. 

Neither the name of the National University of University nor the names of its contributors
may be used to endorse or promote products derived from this software without specific
prior written permission from the National University of Singapore. 

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO THE IMPLIED WARRANTIES 
OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT
SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED
TO, PROCUREMENT OF SUBSTITUTE  GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
DAMAGE.

*/

#include <hip/device_functions.h>

#include "pba2D.h"

// Parameters for CUDA kernel executions
#define BLOCKX		16
#define BLOCKY		16
#define BLOCKSIZE	128
#define TILE_DIM	32
#define BLOCK_ROWS	8

/****** Global Variables *******/
short2 **pbaTextures;       // Two textures used to compute 2D Voronoi Diagram

int pbaMemSize;             // Size (in bytes) of a texture
int pbaTexSize;             // Texture size (squared texture)

texture<short2> pbaTexColor; 
texture<short2> pbaTexLinks; 

/********* Kernels ********/
#include "pba2DKernel.h"

///////////////////////////////////////////////////////////////////////////
//
// Initialize necessary memory for 2D Voronoi Diagram computation
// - textureSize: The size of the Discrete Voronoi Diagram (width = height)
//
///////////////////////////////////////////////////////////////////////////
void pba2DInitialization(int textureSize)
{
  pbaTexSize = textureSize; 
  pbaMemSize = pbaTexSize * pbaTexSize * sizeof(short2); 
  
  pbaTextures = (short2 **) malloc(2 * sizeof(short2 *)); 
  
  // Allocate 2 textures
  hipMalloc((void **) &pbaTextures[0], pbaMemSize); 
  hipMalloc((void **) &pbaTextures[1], pbaMemSize); 
}

///////////////////////////////////////////////////////////////////////////
//
// Deallocate all allocated memory
//
///////////////////////////////////////////////////////////////////////////
void pba2DDeinitialization()
{
    hipFree(pbaTextures[0]); 
    hipFree(pbaTextures[1]); 

    free(pbaTextures); 
}

// Copy input to GPU 
void pba2DInitializeInput(short *input)
{
    hipMemcpy(pbaTextures[0], input, pbaMemSize, hipMemcpyHostToDevice); 
}

void pba2DInitializeInput_d(short *input)
{
    hipMemcpy(pbaTextures[0], input, pbaMemSize, hipMemcpyDeviceToDevice); 
}

// In-place transpose a squared texture. 
// Block orders are modified to optimize memory access. 
// Point coordinates are also swapped. 
void pba2DTranspose(short2 *texture)
{
    dim3 block(TILE_DIM, BLOCK_ROWS); 
    dim3 grid(pbaTexSize / TILE_DIM, pbaTexSize / TILE_DIM); 

    hipBindTexture(0, pbaTexColor, texture); 
    kernelTranspose<<< grid, block >>>(texture, pbaTexSize); 
    hipUnbindTexture(pbaTexColor); 
}

// Phase 1 of PBA. m1 must divides texture size
void pba2DPhase1(int m1) 
{
    dim3 block = dim3(BLOCKSIZE);   
    dim3 grid = dim3(pbaTexSize / block.x, m1); 

    // Flood vertically in their own bands
    hipBindTexture(0, pbaTexColor, pbaTextures[0]); 
    kernelFloodDown<<< grid, block >>>(pbaTextures[1], pbaTexSize, pbaTexSize / m1); 
    hipUnbindTexture(pbaTexColor); 

    hipBindTexture(0, pbaTexColor, pbaTextures[1]); 
    kernelFloodUp<<< grid, block >>>(pbaTextures[1], pbaTexSize, pbaTexSize / m1); 

    // Passing information between bands
    grid = dim3(pbaTexSize / block.x, m1); 
    kernelPropagateInterband<<< grid, block >>>(pbaTextures[0], pbaTexSize, pbaTexSize / m1); 

    hipBindTexture(0, pbaTexLinks, pbaTextures[0]); 
    kernelUpdateVertical<<< grid, block >>>(pbaTextures[1], pbaTexSize, m1, pbaTexSize / m1); 
    hipUnbindTexture(pbaTexLinks); 
    hipUnbindTexture(pbaTexColor); 
}

// Phase 2 of PBA. m2 must divides texture size
void pba2DPhase2(int m2) 
{
    // Compute proximate points locally in each band
    dim3 block = dim3(BLOCKSIZE);   
    dim3 grid = dim3(pbaTexSize / block.x, m2); 
    hipBindTexture(0, pbaTexColor, pbaTextures[1]); 
    kernelProximatePoints<<< grid, block >>>(pbaTextures[0], pbaTexSize, pbaTexSize / m2); 

    hipBindTexture(0, pbaTexLinks, pbaTextures[0]); 
    kernelCreateForwardPointers<<< grid, block >>>(pbaTextures[0], pbaTexSize, pbaTexSize / m2); 

    // Repeatly merging two bands into one
    for (int noBand = m2; noBand > 1; noBand /= 2) {
        grid = dim3(pbaTexSize / block.x, noBand / 2); 
        kernelMergeBands<<< grid, block >>>(pbaTextures[0], pbaTexSize, pbaTexSize / noBand); 
    }

    // Replace the forward link with the X coordinate of the seed to remove
    // the need of looking at the other texture. We need it for coloring.
    grid = dim3(pbaTexSize / block.x, pbaTexSize); 
    kernelDoubleToSingleList<<< grid, block >>>(pbaTextures[0], pbaTexSize); 
    hipUnbindTexture(pbaTexLinks); 
    hipUnbindTexture(pbaTexColor); 
}

#include <stdio.h>

// Phase 3 of PBA. m3 must divides texture size
void pba2DPhase3(int m3) 
{
  dim3 block = dim3(BLOCKSIZE / m3, m3); 
  dim3 grid = dim3(pbaTexSize / block.x); 
  hipBindTexture(0, pbaTexColor, pbaTextures[0]); 
  kernelColor<<< grid, block >>>(pbaTextures[1], pbaTexSize); 
  hipUnbindTexture(pbaTexColor); 
}

void pba2DCompute(int floodBand, int maurerBand, int colorBand)
{
    // Vertical sweep
    pba2DPhase1(floodBand); 

    pba2DTranspose(pbaTextures[1]); 

    // Horizontal coloring
    pba2DPhase2(maurerBand); 

    // Color the rows. 
    pba2DPhase3(colorBand); 

    pba2DTranspose(pbaTextures[1]); 
}

// Compute 2D Voronoi diagram
// Input: a 2D texture. Each pixel is represented as two "short" integer. 
//    For each site at (x, y), the pixel at coordinate (x, y) should contain 
//    the pair (x, y). Pixels that are not sites should contain the pair (MARKER, MARKER)
// See original paper for the effect of the three parameters: 
//    phase1Band, phase2Band, phase3Band
// Parameters must divide textureSize
void pba2DVoronoiDiagram(short *input, short *output, int floodBand, int maurerBand, int colorBand) 
{
    // Initialization
    pba2DInitializeInput(input); 

    // Computation
    pba2DCompute(floodBand, maurerBand, colorBand); 

    // Copy back the result
    hipMemcpy(output, pbaTextures[1], pbaMemSize, hipMemcpyDeviceToHost); 
}


// Added by Kevin Mooney on the 27/08/16
// Transfers the voronoi diagram from input and output arrays on device
void pba2DVoronoiDiagram_d(short *input, short *output, int floodBand, int maurerBand, int colorBand) 
{
    // Initialization
    pba2DInitializeInput_d(input); 

    // Computation
    pba2DCompute(floodBand, maurerBand, colorBand); 

    // Copy back the result
    hipMemcpy(output, pbaTextures[1], pbaMemSize, hipMemcpyDeviceToDevice); 
}

