#include "hip/hip_runtime.h"
/***************************************************
 * Name    : cudaHull.cu
 * Author  : Kevin Mooney
 * Created : 01/07/16
 * Updated : 04/07/16
 *
 * Description:
 *   Monotone chains implementation using thrust
 * 
 * TODO:
 *   IMPLEMENT TEMPLATES in geometry, point and here!
 *
 * NOTES:
 *   This code could drastically improve in readabilty 
 *   if a class was used to store the chains
 ******************************************************/

#include <thrust/adjacent_difference.h>
#include <thrust/copy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/remove.h>
#include <thrust/sort.h>



// Jose doesn't use vector at all...
#include <iostream>
#include <vector>

#include "cudaHull.hpp"
#include "geometry.hpp"

typedef thrust::tuple<int,float,float,bool> id_coord_rotation_tuple; 
typedef thrust::tuple<int,float,float> id_coord_tuple; 
typedef thrust::tuple<float,float> coord_tuple; 

// functor for comparing x-value of float2s
struct compareX : public thrust::binary_function<id_coord_tuple,id_coord_tuple,bool> { 
  __host__ __device__
  bool operator()(const id_coord_tuple &a, const id_coord_tuple &b) {
    return a.get<1>() < b.get<1>();
  }
};

// Returns whether the cross product between two vectors is positive
struct rotation : public thrust::binary_function<coord_tuple,coord_tuple,bool> { 
  __host__ __device__
  bool operator() ( const coord_tuple &a, const coord_tuple &b) {
    return ( a.get<1>()*b.get<0>() - a.get<0>()*b.get<1>() ) > 0;
  }
};

// Upper Chain
// Determines if point currently makes a convex angle with neighbouring points
struct upwardFacing : public thrust::unary_function<id_coord_rotation_tuple,bool> { 
  __host__ __device__
  bool operator() ( const id_coord_rotation_tuple &a) {
    return !a.get<3>();
    // return true;
  }
};

// Lower Chain
// Determines if point currently makes a convex angle with neighbouring points
struct downwardFacing : public thrust::unary_function<id_coord_rotation_tuple,bool> { 
  __host__ __device__
  bool operator() ( const id_coord_rotation_tuple &a) {
    return a.get<3>();
    // return true;
  }
};

// Not needed anymore, only for reference
// functor for adding float2s
struct add_float2 : public thrust::binary_function<float2,float2,float2> { 
  __host__ __device__
  float2 operator()(const float2 &a, const float2 &b) {
    float2 r;
    r.x = a.x + b.x;
    r.y = a.y + b.y;
    return r;
  }
};

// functor for shifting a float2
struct shift_float2 : public thrust::unary_function<float2,void> { 
  float2 shift;
  shift_float2( const float2 &_shift ) {
    shift.x = _shift.x;
    shift.y = _shift.y;
  }

  __host__ __device__
  void operator()( float2 &a ) {
    a.x -= shift.x;
    a.y -= shift.y;
  }
};

struct angle_float2 : public thrust::unary_function<float2,float> {
  __host__ __device__
  float operator() ( const float2 &a ) {
    return atan2 ( a.y, a.x );
  }
};

std::vector< size_t > cudaHull ( const CompGeom::Geometry &geom ) {

  // Transfer geometry to host vector
  thrust::host_vector<float> h_xvec(geom.size());
  thrust::host_vector<float> h_yvec(geom.size());
  for ( size_t i=0; i<geom.size(); i++ ) {
    h_xvec[i] = geom[i][0];
    h_yvec[i] = geom[i][1];
  }

  // transfer data to the device                                                                  
  thrust::device_vector<float> d_xvec = h_xvec;
  thrust::device_vector<float> d_yvec = h_yvec;

  // Fill an array of IDS on device
  thrust::device_vector<int> ids(d_xvec.size());
  thrust::sequence(ids.begin(),ids.end());

  // Find average coordinate to be new origin
  float avex, avey;
  avex = thrust::reduce ( d_xvec.begin(), d_xvec.end() ) / d_xvec.size();
  avey = thrust::reduce ( d_yvec.begin(), d_yvec.end() ) / d_yvec.size();

  // Make constant iterators with the shift value
  thrust::constant_iterator<int> shift_x(avex);
  thrust::constant_iterator<int> shift_y(avey);

  // // translate through average coordinate making it the new origin
  thrust::transform ( d_xvec.begin(), d_xvec.end(), shift_x, d_xvec.begin(), thrust::plus<float>() );
  thrust::transform ( d_yvec.begin(), d_yvec.end(), shift_y, d_yvec.begin(), thrust::plus<float>() );

  // Zip IDS and coordinates
  auto zip_start = make_zip_iterator(make_tuple(ids.begin(), 
  					    d_xvec.begin(),
					    d_yvec.begin()));
  auto zip_end   = make_zip_iterator(make_tuple(ids.end(), 
  					    d_xvec.end(),
					    d_yvec.end()));
  // Sort by x-coordinate
  thrust::sort(zip_start, zip_end, compareX());

  // Copy ids and vectors into lower chain
  thrust::device_vector<int> lower_ids(ids.begin()   ,ids.end()   );
  thrust::device_vector<float> lower_x(d_xvec.begin(),d_xvec.end());
  thrust::device_vector<float> lower_y(d_yvec.begin(),d_yvec.end());

  // Calculate the vectors between each point
  thrust::device_vector<float> vdiff_x(ids.size());
  thrust::device_vector<float> vdiff_y(ids.size());
  thrust::adjacent_difference(d_xvec.begin(),d_xvec.end(),vdiff_x.begin());
  thrust::adjacent_difference(d_yvec.begin(),d_yvec.end(),vdiff_y.begin());

  // Determine if the angle between consecutive vectors is convex up
  // End vectors are always true
  thrust::device_vector<bool> upper_convex(ids.size());
  thrust::fill(upper_convex.begin(),upper_convex.end(),true);
  auto diff_start = make_zip_iterator ( make_tuple ( vdiff_x.begin(), vdiff_y.begin() ) );
  auto diff_end   = make_zip_iterator ( make_tuple ( vdiff_x.end()  , vdiff_y.end()   ) );  
  thrust::transform ( diff_start+2, diff_end, diff_start+1, upper_convex.begin()+1, rotation() );
  
  // Remove downward facing points
  bool * upper_convex_ptr = thrust::raw_pointer_cast(upper_convex.data());
  auto four_tuple_start = make_zip_iterator ( make_tuple ( ids.begin(),
							   d_xvec.begin(),
							   d_yvec.begin(),
							   upper_convex.begin()));
  auto four_tuple_end   = make_zip_iterator ( make_tuple ( ids.end(),
							   d_xvec.end(),
							   d_yvec.end(),
							   upper_convex.end()));
  auto new_end = thrust::remove_if ( four_tuple_start, four_tuple_end, upwardFacing() ); 

  // auto a = new_end.get_iterator_tuple();

  int old_N = 0;
  int N = thrust::distance (four_tuple_start, new_end );
  
  while ( old_N != N ) {
    thrust::adjacent_difference(d_xvec.begin(),d_xvec.begin() + N ,vdiff_x.begin());
    thrust::adjacent_difference(d_yvec.begin(),d_yvec.begin() + N ,vdiff_y.begin());
    
    thrust::transform ( diff_start+2, diff_start + N , diff_start+1, upper_convex.begin()+1, rotation() );
    
    new_end = thrust::remove_if ( four_tuple_start, four_tuple_start + N, upwardFacing() ); 
    
    old_N = N;
    N = thrust::distance (four_tuple_start, new_end );
  }

  four_tuple_start = make_zip_iterator ( make_tuple ( lower_ids.begin(),
						      lower_x.begin(),
						      lower_y.begin(),
						      upper_convex.begin()));
  four_tuple_end   = make_zip_iterator ( make_tuple ( lower_ids.end(),
						      lower_x.end(),
						      lower_y.end(),
						      upper_convex.end()));
  
  // four_tuple_start = make_zip_iterator ( make_tuple ( ids.begin(),
  // 							   d_xvec.begin(),
  // 							   d_yvec.begin(),
  // 							   upper_convex.begin()));

  // LOWER CHAIN
  thrust::adjacent_difference(lower_x.begin(),lower_x.end(),vdiff_x.begin());
  thrust::adjacent_difference(lower_y.begin(),lower_y.end(),vdiff_y.begin());
  thrust::fill(upper_convex.begin(),upper_convex.end(),false);
  thrust::transform ( diff_start+2, diff_end, diff_start+1, upper_convex.begin()+1, rotation() );
  new_end = thrust::remove_if ( four_tuple_start, four_tuple_end, downwardFacing() ); 

  int upper_N = N; 		// Need a better naming convention
  old_N = 0;
  N = thrust::distance (four_tuple_start, new_end );
  while ( old_N != N ) {
    thrust::adjacent_difference(lower_x.begin(),lower_x.begin() + N ,vdiff_x.begin());
    thrust::adjacent_difference(lower_y.begin(),lower_y.begin() + N ,vdiff_y.begin());

    thrust::transform ( diff_start+2, diff_start + N , diff_start+1, upper_convex.begin()+1, rotation() );

    new_end = thrust::remove_if ( four_tuple_start, four_tuple_start + N, downwardFacing() );

    old_N = N;
    N = thrust::distance (four_tuple_start, new_end );
  }

  // for ( size_t i=0; i<upper_N; i++ ) {
  //   std::cout << ids[i] << std::endl; 
  // }
  // for ( size_t i=N-1; i--; ) {
  //   std::cout << lower_ids[i] << std::endl; 
  // }  

  // std::cout << *(new_end.get<0>()) << std::endl;

  // auto lower_start = make_zip_iterator(make_tuple(lower_ids.begin(), 
  // 					    lower_x.begin(),
  // 					    lower_y.begin()));
  // auto lower_end   = make_zip_iterator(make_tuple(ids.end(), 
  // 					    lower_x.end(),
  // 					    lower_y.end()));  

  // Copy back to host
  std::vector< size_t > result (N + upper_N - 1);
  thrust::copy(ids.begin(), ids.begin()+upper_N, result.begin());
  thrust::copy(lower_ids.rend()-N + 1, lower_ids.rend(), result.begin()+upper_N);
  // for ( size_t i=0; i<ids.size(); i++ ) {
  //   std::cout << ids[i] << " " << d_xvec[i] << " " << d_yvec[i] << "\t";
  //   std::cout << lower_ids[i] << " " << lower_x[i] << " " << lower_y[i] << "\t" 
  // 	      << vdiff_x[i] << " " << vdiff_y[i] << " " << upper_convex[i] <<  std::endl;
  // }
  // To surpress compiler warnings, not a legitimate strategy
  return result;
}
