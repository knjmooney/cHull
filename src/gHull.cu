#include "hip/hip_runtime.h"
/******************************************************
 * Name    : gHull.hup
 * Author  : Kevin Mooney
 * Created : 26/08/16
 * Updated :
 *
 * Description:
 *
 * NOTES:
 *   Class ideas were taken from cuda quadtree sample
 ******************************************************/

#include <iostream>
#include <limits>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/tuple.h>
#include <thrust/unique.h>
#include <string>

#include "chooseCard.cuh"
#include "directionEnums.hpp"
#include "errorMessages.hpp"
#include "geometry.hpp"
#include "gHull.cuh"
#include "pba2D.h"
#include "voronoi.hpp"

#define XBLOCKSIZE  32
#define YBLOCKSIZE  32

#define RES         512		// Resolution of digital projection

#define NFACES      6		// Number of faces of a cube...
#define DIM         3		// Dimensions
#define EPS         1e-5	// Epsilon
#define MAXSTARSIZE 10

// pba parameters, choice of parameters discussed in pba paper
// These are important, they determine the blocksizes
#define P1B	16		// Phase 1 band
#define P2B	16		// Phase 2 band
#define P3B	16		// Phase 3 band


// typedef thrust::device_vector dvec;
// typedef thrust::host_vector   hvec;
template <typename T> using dvec = thrust::device_vector<T>;
template <typename T> using hvec = thrust::host_vector<T>;

// Decided against namespace std becomes too many conflicts with thrust
using std::cout;
using std::endl;

// Trick to dynamically access float 3's
union float_type
{
  float3 data3;
  float data[3];
};


// Class for improving contiguous memory accessing
class Points {
  float *_x;
  float *_y;
  float *_z;

public:
  __host__ __device__ 
  Points() : _x(NULL), _y(NULL), _z(NULL) {};

  __host__ __device__ 
  Points(float *x, float *y, float *z) : _x(x), _y(y), _z(z) {};

  __host__ __device__ __forceinline__
  float3 get_point(int id) const {
    return make_float3(_x[id],_y[id],_z[id]);
  }

  __host__ __device__ __forceinline__ 
  void set_point(int id, const float3 &p) {
    _x[id] = p.x;
    _y[id] = p.y;
    _z[id] = p.z;
  }

  // Set the pointers. 
  __host__ __device__ __forceinline__ 
  void set(float *x, float *y, float *z) {
    _x = x;
    _y = y;
    _z = z;
  }
};

// Class for storing projections and IDS
class Face {
  float *data;
  int *pids;
  int length;

  __host__ __device__ __forceinline__
  int index(const int i, const int j) const {return j + i*length; }
public:
  __host__ __device__
  Face() : data{NULL}, pids{NULL}, length{0} {}
  
  __host__ __device__
  Face( float *data, int *ids, int length ) : data{data}, pids{ids}, length{length} {} 

  __host__ __device__ __forceinline__
  void set ( float *datas, int *ids, int L ) { data = datas; pids = ids; length = L; }
  
  __host__ __device__ __forceinline__
  float get_data ( int idx, int idy ) const { return data[index(idx,idy)]; }
  
  __host__ __device__ __forceinline__
  void set_data (int idx, int idy, float val) {  data[index(idx,idy)] = val; }
  
  __host__ __device__ __forceinline__
  float get_id ( int idx,int idy ) const { return pids[index(idx,idy)]; }
  
  // Value is the point id and id is the index
  __host__ __device__ __forceinline__
  void set_id (int idx,int idy, int val) {  pids[index(idx,idy)] = val; }  
  
  __host__ __device__ __forceinline__
  int get_length() { return length; }
};


// Class for improving contiguous memory accessing
class Stars {
  int *_ids;
  int *_edgeList;
  int *_sizes;
  int _nstars;
  int _maxsize;
  //  bool alive; Does size not have this role?

public:
  __host__ __device__ 
  Stars() : _ids(NULL), _edgeList(NULL), _sizes(NULL), _nstars(0), _maxsize(0) {};

  __host__ __device__ 
  Stars(int *ids, int *edgeList, int *sizes, int nstar, int maxsize ) :
    _ids(ids), _edgeList(edgeList), _sizes(sizes), _nstars(nstar), _maxsize(maxsize) {};

  // Set the pointers.                
  __host__ __device__ __forceinline__ 
  void set(int *ids, int *edgeList, int *sizes, int nstar, int maxsize ) {
    _ids = ids;
    _edgeList = edgeList;
    _sizes = sizes;
    _nstars = nstar;
    _maxsize = maxsize;
  }

  __host__ __device__ __forceinline__
  int get_id(int i) const {
    return _ids[i];
  }

  __host__ __device__ __forceinline__
  int *get_edgeList(int i) {
    return &_edgeList[_maxsize*i];
  }

  __host__ __device__ __forceinline__
  int get_size ( int i ) const {
    return _sizes[i];
  }
  
  __host__ __device__ __forceinline__
  void set_size ( int i, int sz ) {
    if ( sz > _maxsize ) { printf ( "STAR IS TOO BIG\n" ); asm("trap;"); }
    _sizes[i] = sz;
  }
  
  __host__ __device__ __forceinline__
  int nstars() const {
    return _nstars;
  }
};


template < typename vec > 
void fillHostArrays ( vec &px, vec &py, vec &pz, const CompGeom::Geometry & geom ) {
  size_t i=0;
  for ( const auto &p : geom ) {
    px[i] = p[0];
    py[i] = p[1];
    pz[i] = p[2];
    i++;
  }
}

// Debug kernel
__global__ 
void printPointsOnCard ( const Points *points, const int n_points ) {
  int id = threadIdx.x + blockIdx.x*blockDim.x;

  if ( id < n_points ) {
    float3 myp = points->get_point(id);
    printf ( "%d: %lf %lf %lf\n", id, myp.x, myp.y, myp.z ); 
  }
}

// Finds extrema on the device
// If the compiler is smart then the dereference won't transfer to host and then back
void findExtremes_d ( dvec<float> &extrm_d, 
		      const dvec<float> &px_d, 
		      const dvec<float> &py_d, 
		      const dvec<float> &pz_d ) 
{
  auto extx = thrust::minmax_element(px_d.begin(),px_d.end());
  auto exty = thrust::minmax_element(py_d.begin(),py_d.end());
  auto extz = thrust::minmax_element(pz_d.begin(),pz_d.end());
  extrm_d[Direction::LEFT ] = *extx.first;
  extrm_d[Direction::RIGHT] = *extx.second;
  extrm_d[Direction::BACK ] = *exty.first;
  extrm_d[Direction::FRONT] = *exty.second;
  extrm_d[Direction::DOWN ] = *extz.first;
  extrm_d[Direction::UP   ] = *extz.second;
}

__global__
void projectToFace_d ( Face *face, Points *points, float *ex, int dir, int N ) {
  const int w = face->get_length();
  const int i = dir %DIM;
  const int j = (i+1) %DIM;
  const int k = (i+2) %DIM;
  const float minw = ex[j];
  const float minh = ex[k];
  const float maxw = ex[j+DIM];
  const float maxh = ex[k+DIM];
  const float pos  = ex[dir];

  const int id = threadIdx.x + blockIdx.x*blockDim.x;
  
  // printf ( "%d\n", id );

  if ( id < N )  {
    float_type p;
    p.data3 = points->get_point(id);
    
    int idj = int(w*(p.data[j]-minw)/((maxw-minw)*(1+EPS)));
    int idk = int(w*(p.data[k]-minh)/((maxh-minh)*(1+EPS)));

    float       cmin_val = face->get_data(idj,idk);
    float       this_min = fabs(p.data[i]-pos);

    if ( cmin_val > this_min ) {
      face->set_data  (idj,idk, this_min   );
      face->set_id    (idj,idk, id         );
    }
  }
}

// Each thread spawns a projection kernel (dynamic parallelism 
__global__
void projectToBox_d ( Face *face_d, Points *points_d, float *extrm_d, int N ) {
  dim3 dimBlock ( XBLOCKSIZE );
  dim3 dimGrid ( (N/dimBlock.x) + (!(N%dimBlock.x)?0:1) );
  int dir = threadIdx.x;
  if ( dir < NFACES ) {
    projectToFace_d <<<dimGrid,dimBlock>>>( face_d+dir, points_d, extrm_d, dir, N );
  }
}

__host__ __device__
inline int shortID ( int i, int j ) {
  return 2*(i*RES + j );
}

// pba transposes the coordinates, hence idx and idy are swapped at assignment
__global__
void fillInput_d(short * input, Face * face ) {
  int idx   = blockIdx.x*blockDim.x+threadIdx.x;
  int idy   = blockIdx.y*blockDim.y+threadIdx.y;
  
  if ( idx < RES && idy < RES ) {
    if ( face->get_id(idx,idy) != MARKER ) {
      int index = shortID ( idx, idy );
      input[index]   = idy;
      input[index+1] = idx;
    }
  }
}

// Range based loops in CUDA
__global__
void countOutEdges_d(int *out_edges_p,short *V) {
  int idx   = blockIdx.x*blockDim.x+threadIdx.x;
  int idy   = blockIdx.y*blockDim.y+threadIdx.y;

  if ( idx < RES && idy < RES ) {
    int index  = idx*RES + idy;
    int sindex = 2*index;
    // int count  = 0;

    const int dir[4][2] =  {{-1,0},{0 ,-1},{0 ,1},{1 ,0}};
    for ( auto d : dir ) {
      int ni = idx+d[0];
      int nj = idy+d[1];
      
      int nindex = shortID(ni,nj);
      // Check bounds first 
      if ( !(ni < 0 || nj < 0 || ni >= RES || nj >= RES) 
      	   && (V[nindex] != V[sindex] || V[nindex+1] != V[sindex+1]) )
      	{
	  // count++;
	  out_edges_p[index]++;
	}
    }
    // out_edges_p[index] = max(0,count-2);
  }  
}

__global__
void constructWorkingSet_d(int *frsts,int*scnds,short *V, Face *face,int *out_edges_offest) {
  int idx   = blockIdx.x*blockDim.x+threadIdx.x;
  int idy   = blockIdx.y*blockDim.y+threadIdx.y;

  if ( idx < RES && idy < RES ) {
    int index  = idx*RES + idy;
    int sindex = 2*index;
    int count  = 0;

    const int dir[4][2] =  {{-1,0},{0 ,-1},{0 ,1},{1 ,0}};
    for ( auto d : dir ) {
      int ni = idx+d[0];
      int nj = idy+d[1];
      
      int nindex = shortID(ni,nj);
      // Check bounds first               
      if ( !(ni < 0 || nj < 0 || ni >= RES || nj >= RES) 
      	   && (V[nindex] != V[sindex] || V[nindex+1] != V[sindex+1]) )
      	{
	  frsts[out_edges_offest[index]+count] = face->get_id ( V[sindex+1], V[sindex] );   
	  scnds[out_edges_offest[index]+count] = face->get_id ( V[nindex+1], V[nindex] );   
	  count++;
	}
    }
    // If only one edge was added, then it's definitely double counting
    // if ( count == 1 ) {
    //   frsts[index*4] = frsts[index*4+1];   
    //   scnds[index*4] = scnds[index*4+1];   
    // }
  }  
}

// predicate functor for comparing edges on device
typedef thrust::tuple<int,int> edgeTuple;

struct compareEdges : public thrust::binary_function<edgeTuple,edgeTuple,bool> {
  __host__ __device__
  bool operator()(const edgeTuple &a, const edgeTuple &b) {
    return a.get<0>() != b.get<0>() ? a.get<0>() < b.get<0>() : a.get<1>() < b.get<1>();
  }
};


void constructVoronois_d( dvec<int> &firsts, dvec<int> &seconds, Face *face ) {

  const size_t RSQ = RES*RES;

  dvec<short> input ( 2*RSQ );
  std::vector < dvec<short> > output (3,dvec<short>(2*RSQ));

  dim3 dimBlock ( XBLOCKSIZE, YBLOCKSIZE  );
  dim3 dimGrid  ( (RES/dimBlock.x) + (!(RES%dimBlock.x)?0:1), 
		  (RES/dimBlock.y) + (!(RES%dimBlock.y)?0:1) );

  // Initialise memory for PBA algorithm on device
  pba2DInitialization(RES);

  for ( int i=0; i<3; i++ ) {
    short * input_p  = thrust::raw_pointer_cast(&input[0]) ;
    short * output_p = thrust::raw_pointer_cast(&output[i][0]);

    thrust::fill(input.begin(),input.end(),MARKER );
    fillInput_d <<< dimGrid,dimBlock >>> (input_p,face+i );
    // Don't know what the last three numbers do
    pba2DVoronoiDiagram_d(input_p,output_p,P1B,P2B,P3B); 

    hvec<short> input_h = input, output_h = output[i];
    makeVoronoiPBM ( &output_h[0], &input_h[0], RES,RES, 
		     "parallel" + std::to_string(i) + ".pbm" );
  }

  // Free PBA memory
  pba2DDeinitialization();

  /////////// This was an attempt at finding the dual using a prefix sum

  dvec<int> n_out_edges(6*RSQ,0);
  for ( int i=0; i<3; i++ ) {
    short * output_p    = thrust::raw_pointer_cast(&output[i][0]);
    int   * out_edges_p;

    out_edges_p = thrust::raw_pointer_cast(&n_out_edges[i*RSQ]);
    countOutEdges_d<<<dimBlock,dimGrid>>>(out_edges_p,output_p);

    out_edges_p = thrust::raw_pointer_cast(&n_out_edges[(i+DIM)*RSQ]);
    countOutEdges_d<<<dimBlock,dimGrid>>>(out_edges_p,output_p);    
  }

  //Find the offsets for each array plus the output size
  size_t sz = n_out_edges.back();
  thrust::exclusive_scan(n_out_edges.begin(),n_out_edges.end(),n_out_edges.begin());
  sz += n_out_edges.back();

  // cout << "Size of initial triangulation: " << sz << endl; 
  firsts  = dvec< int > (sz,std::numeric_limits<int>::max());
  seconds = dvec< int > (sz,std::numeric_limits<int>::max());

  // for ( int i=0; i<3; i++ ) {
  //   short * output_p    = thrust::raw_pointer_cast(&output[i][0]);
  //   int   * firsts_p    = thrust::raw_pointer_cast(&firsts[0]   );
  //   int   * seconds_p   = thrust::raw_pointer_cast(&seconds[0] );
  //   int   * out_edges_p;

  //   out_edges_p = thrust::raw_pointer_cast(&n_out_edges[i*RSQ]);
  //   // countOutEdges_d<<<dimBlock,dimGrid>>>(out_edges_p,output_p);

  //   out_edges_p = thrust::raw_pointer_cast(&n_out_edges[(i+DIM)*RSQ]);
  //   // countOutEdges_d<<<dimBlock,dimGrid>>>(out_edges_p,output_p);    
  // }

  //////////////////////////////// CONSTRUCT WORKING SETS ////////////////////////////////

  //////// I just allocated the maximum possible need memory and then reduce it at the end

  // firsts  = dvec<int>(6*4*RES*RES,std::numeric_limits<int>::max());  
  // seconds = dvec<int>(6*4*RES*RES,std::numeric_limits<int>::max());  
  for ( int i=0; i<3; i++ ) {
    short * output_p    = thrust::raw_pointer_cast(&output[i][0]);
    int   * firsts_p;
    int   * seconds_p;
    int   * out_edges_p;

    // All the min faces
    firsts_p    = thrust::raw_pointer_cast(&firsts[0]   );
    seconds_p   = thrust::raw_pointer_cast(&seconds[0]  );
    out_edges_p = thrust::raw_pointer_cast(&n_out_edges[i*RSQ]);
    constructWorkingSet_d<<<dimBlock,dimGrid>>> (firsts_p,
						 seconds_p,
						 output_p,
						 &face[i],
						 out_edges_p);

    // All the max faces
    // firsts_p    = thrust::raw_pointer_cast(&firsts[(i+DIM)*4*RSQ]   );
    // seconds_p   = thrust::raw_pointer_cast(&seconds[(i+DIM)*4*RSQ]  );
    out_edges_p = thrust::raw_pointer_cast(&n_out_edges[(i+DIM)*RSQ]);
    constructWorkingSet_d<<<dimBlock,dimGrid>>> (firsts_p,
						 seconds_p,
						 output_p,
						 &face[i+DIM],
						 out_edges_p);
  }  
  
  //////////////////// SORT AND REMOVE DUPLICATES ////////////////////////////////

  auto edgesStart 
    = thrust::make_zip_iterator ( thrust::make_tuple ( firsts.begin(), 
						       seconds.begin() ) );
  auto edgesEnd   
    = thrust::make_zip_iterator ( thrust::make_tuple ( firsts.end(),   
						       seconds.end()   ) );

  // cout << "Before Removing Markers      : " 
  //      << thrust::distance ( edgesStart,edgesEnd ) << endl;
  // edgesEnd = thrust::remove ( edgesStart, edgesEnd, 
  // 			      thrust::tuple<int,int> (std::numeric_limits<int>::max(),
  // 						      std::numeric_limits<int>::max()));

  // Remove some duplicates before sorting
  // cout << "Before Removing Duplicates 1 : " 
  //      << thrust::distance ( edgesStart,edgesEnd ) << endl;
  // edgesEnd = thrust::unique ( edgesStart, edgesEnd );
  thrust::sort ( edgesStart, edgesEnd  , compareEdges() );
  // cout << "Before Removing Duplicates 2 : " 
  //      << thrust::distance ( edgesStart,edgesEnd ) << endl;
  edgesEnd = thrust::unique ( edgesStart, edgesEnd );

  firsts.resize(thrust::distance ( edgesStart,edgesEnd ));
  seconds.resize(thrust::distance ( edgesStart,edgesEnd ));

  // cout << "Final Size : " << firsts.size() << endl;
  // dvec<int> star_count
  // for ( auto it = edgesStart; it != edgesEnd; ++it ) {
  //   cout << thrust::get<0>(*it) << " " << thrust::get<1>(*it) << endl;
  // }
}

// Overload the difference operator for float3
__device__
float3 operator-(float3 a, float3 b) {
  return make_float3 ( a.x - b.x, a.y - b.y, a.z - b.z );
}

// Calculates the normal vector of a float3
__device__
inline float3 norm ( float3 tri[3] ) {
  float3 a = tri[1] - tri[0], b = tri[2] - tri[0]; 
  return make_float3 ( a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.y, a.x*b.y - a.y*b.x );
}

// Determines if p is visible to tri
__device__
inline bool isVisible (float3 tri[3], float3 p) {
  float3 n   = norm ( tri );
  // float3 com = average(tri);
  // float3 v = com - p;
  // dot product
  return true;
}

__global__
void constructStars_Kernel ( Stars *stars, 
			     int* seconds, 
			     int * scnds_offsets, 
			     Points *points ) 
{ 
  int idx   = blockIdx.x*blockDim.x+threadIdx.x;

  if ( idx < stars->nstars() ) {
    int  myid = stars->get_id(idx);
    int *myws = &seconds[scnds_offsets[idx]];
    int  mysz = stars->get_size(idx);
    int *myel = stars->get_edgeList(idx);
    
    float3 tri[3];
    tri[0]   = points->get_point(myid);
    tri[1]   = points->get_point(myws[0]);
    tri[2]   = points->get_point(myws[1]);

    float3 p = points->get_point(myws[2]);

    if ( isVisible ( tri, p ) ) 
      ;; //Then invert

  }
}

static void constructStars_d ( Stars *stars_d, 
			       dvec<int> &seconds, 
			       dvec<int> &scnds_offsets, 
			       Points *points_d )
{
  int n_stars = scnds_offsets.size();

  dim3 dimBlock ( XBLOCKSIZE );
  dim3 dimGrid ( (n_stars/dimBlock.x) + (!(n_stars%dimBlock.x)?0:1) );  

  constructStars_Kernel <<< dimGrid, dimBlock >>> 
    ( stars_d,
      thrust::raw_pointer_cast(&seconds[0]),
      thrust::raw_pointer_cast(&scnds_offsets[0]),
      points_d );
}


std::vector < std::vector < size_t > > gHull ( const CompGeom::Geometry &geom ) {

  // chooseCudaCard(0);
  gpuErrchk ( hipGetLastError() );

  size_t N = geom.size();

  /////////////////////////////// INIT POINTS ON DEVICE /////////////////////////////////
  
  // Fill three 1d arrys with the geometry
  hvec<float> px_h(N), py_h(N), pz_h(N);
  fillHostArrays ( px_h, py_h, pz_h , geom   );

  // Transfer points to card
  dvec<float> px_d = px_h, py_d = py_h, pz_d = pz_h;

  //  Initialise geometry class on card
  //  Pass the device vectors rather than the host vectors
  Points points_h (thrust::raw_pointer_cast(&px_d[0]), 
		   thrust::raw_pointer_cast(&py_d[0]), 
		   thrust::raw_pointer_cast(&pz_d[0]) );
  Points *points_d;
  hipMalloc ( (void **) &points_d, sizeof(Points) );
  hipMemcpy ( points_d, &points_h, sizeof(Points), hipMemcpyHostToDevice );

  /////// PRINT POINTS ON CARD //////////////
  // dim3 dimBlock ( XBLOCKSIZE );
  // dim3 dimGrid ( (N/dimBlock.x) + (!(N%dimBlock.x)?0:1) );
  // printPointsOnCard <<<dimGrid,dimBlock>>> ( points_d, N );
  
  ////////////////////////////////// SETUP FACES ////////////////////////////////////

  //Init faces
  // Numric limits may be different on device, be careful!
  std::vector < dvec<float> > datas_d ( NFACES,  dvec<float> (RES*RES,std::numeric_limits<float>::max()) );
  std::vector < dvec<int  > > pids_d  ( NFACES,  dvec<int  > (RES*RES, MARKER) );

  Face face_h[NFACES];
  Face *face_d;
  for ( size_t i=0; i<NFACES; i++ ) {
    face_h[i].set (thrust::raw_pointer_cast(&datas_d[i][0]),
		   thrust::raw_pointer_cast(&pids_d[i][0]), 
		   RES );
  }
  hipMalloc ( (void **) &face_d, NFACES*sizeof(Face) );
  hipMemcpy ( face_d, &face_h, NFACES*sizeof(Face), hipMemcpyHostToDevice );

  ////////////////////////////////// FIND EXTREMA /////////////////////////////////

  // Find mins and maxs
  // Might be faster to do this on the CPU
  dvec<float> extrm_d(6);
  findExtremes_d ( extrm_d, px_d, py_d, pz_d );

  ////////////////////////////////// PROJECTION //////////////////////////////////
  
  projectToBox_d<<<1,NFACES>>> ( face_d, 
				 points_d, 
				 thrust::raw_pointer_cast(&extrm_d[0]), 
				 N );

  ////////////////////////////////// CONSTRUCT VORONOIS //////////////////////////

  // This actually finds all the edges
  dvec< int > firsts;
  dvec< int > seconds;
  constructVoronois_d ( firsts,seconds,face_d );

  ////////////////////////////////// MAKE STARS //////////////////////////

  dvec<int> star_ids   ( N                );
  dvec<int> star_sizes ( firsts.size() ,1 );

  // Find the unique ids and their counts
  auto key_val_end = thrust::reduce_by_key ( firsts.begin()    , 
					     firsts.end()      , 
					     star_sizes.begin(), 
					     star_ids.begin()  ,
					     star_sizes.begin() );

  // for ( auto it = star_ids.begin(), it2 = star_sizes.begin(); 
  // 	it != key_val_end.first; 
  // 	++it, ++it2 ) 
  //   {
  //     cout << *it << ": " << *it2 << endl;
  //   } 
  
  int n_stars = thrust::distance ( star_ids.begin(), key_val_end.first );
  
  dvec<int> edgeList ( MAXSTARSIZE * n_stars ) ;
  dvec<int> scnds_offsets ( n_stars );
  thrust::exclusive_scan ( star_sizes.begin(), star_sizes.begin() + n_stars, scnds_offsets.begin() );
  
  Stars stars_h (thrust::raw_pointer_cast(&star_ids[0]),
		 thrust::raw_pointer_cast(&edgeList[0]),
		 thrust::raw_pointer_cast(&star_sizes[0]),
		 n_stars,
		 MAXSTARSIZE );

  Stars *stars_d;
  hipMalloc ( (void **) &stars_d, sizeof(Stars) );
  hipMemcpy ( stars_d, &stars_h, sizeof(Stars), hipMemcpyHostToDevice );

  constructStars_d ( stars_d, seconds, scnds_offsets, points_d );
  
  ////////////////////////////////// DEBUGGING   /////////////////////////////////
  
  // Transfer back for debugging
  // std::vector < hvec<float> > datas_h(datas_d.begin(),datas_d.end());
  // std::vector < hvec<int  > > pids_h (pids_d.begin(),pids_d.end());
  // for ( auto dat : datas_h ) {
  //   for ( int i=0; i<RES*RES; i++ ) {
  //     if (i%10==0 && i!=0) cout << endl;
  //     cout << dat[i] << " " ;
  //   }
  //   cout << endl << endl;;
  // }

  // for ( auto dat : pids_h ) {
  //   for ( int i=0; i<RES*RES; i++ ) {
  //     if (i%10==0 && i!=0) cout << endl;
  //     cout << dat[i] << " " ;
  //   }
  //   cout << endl << endl;;
  // }

  // Copy back
  // px_h = px_d;
  // py_h = py_d;
  // pz_h = pz_d;

  // for ( size_t i=0; i<px_h.size(); i++ ) {
  //   cout << geom[i] << endl;
  //   cout << px_h[i] << " " << py_h[i] << " " << pz_h[i] << endl << endl;
  // }
  
  hipFree ( points_d );
  hipFree ( face_d  );
  hipFree ( stars_d );
  gpuErrchk ( hipGetLastError() );
  
  return {{1}};
}
